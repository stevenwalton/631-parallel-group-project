#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <vector>
#include <cassert>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_cooperative_groups.h"

#include "cuda_math.h"

// Kernels go up top
//Luis version
//Basically launching one block for each element in the 
//result matrix
template <class T>
__global__ void
cuda_matrix_mult(size_t inner_size, size_t y_cols, T* x, T* y, T* z)
{
    	extern __shared__ double shared_mem[];
    	int row = blockIdx.x;
    	int col = blockIdx.y;
    	int n_threads = blockDim.x;
    	int tid = threadIdx.x;

	size_t i;
    	//clear the shared memory
    	shared_mem[tid] = 0.0;
    	__syncthreads();

	//printf("\nrow = %d col = %d\n", row, col);

	for (i = 0 + tid; i < inner_size; i+=n_threads)
	{
		shared_mem[tid] += x[row * inner_size + i] * y[i * y_cols + col];
	}
	__syncthreads();

	if(tid == 0){
		z[row*y_cols + col] = 0.0;
		for (i = 0; i < n_threads; i++)
			z[row*y_cols + col] += shared_mem[i];
	}
}

//one block per row in the resulting matrix
//each thread will take care of one of the columns
//thus, we need to have as many threads as columns
//Luis' important note: 
//of course this limits the size of the matrix that 
//can actually be obtained to N X 1024 which should
//be enough for our purposes
template <class T>
__global__ void
cuda_matrix_mult_v2(size_t inner_size, T* x, T* y, T* z)
{
        extern __shared__ double shared_mem[];
        int row = blockIdx.x;
        int tid = threadIdx.x;  //basically the col
        int n_cols = blockDim.x;

        size_t i;
        //clear the shared memory
        shared_mem[tid] = 0.0;
        __syncthreads();

        //printf("\nrow = %d col = %d\n", row, col);

        for (i = 0; i < inner_size; ++i)
        {
                shared_mem[tid] += x[row * inner_size + i] * y[i * n_cols + tid];
        }
        __syncthreads();

	//no reduction or further synchronization needed, 
	//simply write to output array
	z[row * n_cols + tid] = shared_mem[tid];
}

/*Steven's version
   template <class T>
__global__ void
cuda_matrix_mult(size_t x_size, size_t y_size,
                 T* x, T* y, T* z)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    if ( row < x_size && col < y_size)
    {
        printf("\nrow = %d col = %d\n", row, col);
        T tmp = 0;
        for (size_t k = 0; k < y_size; ++k)
        {
            tmp += x[row*y_size+ k] * y[k*y_size + col];
            printf("%f * %f\n", x[row*y_size+ k], y[k*y_size + col]);
        }
        z[row*y_size+ col] = tmp;
    }
}
*/
/*
    CopyData function taken from Jee Choi's Homework assignment
 */
template <class T>
void CopyData(
  T* input,
  unsigned int N,
  unsigned int dsize,
  T** d_in)
{
  // timers
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate pinned memory on host (for faster HtoD copy)
  T* h_in_pinned = NULL;
  checkCudaErrors(hipHostMalloc((void**) &h_in_pinned, N * dsize));
  assert(h_in_pinned);
  memcpy(h_in_pinned, input, N * dsize);

  // copy data
  checkCudaErrors(hipMalloc((void**) d_in, N * dsize));
  //checkCudaErrors(hipEventRecord(start, 0));
  checkCudaErrors(hipMemcpy(*d_in, h_in_pinned,
                             N * dsize, hipMemcpyHostToDevice));
  //checkCudaErrors(hipEventRecord(stop, 0));
  //checkCudaErrors(hipEventSynchronize(stop));
  //checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  //printf("  Pinned Device to Host bandwidth (GB/s): %f\n",(N * dsize) * 1e-6 / elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void featuresAndLabelsToGPU(std::vector<std::vector<float>>& features,
                            std::vector<int>& labels,
                            size_t batch_size,
                            float** dev_features,
                            int** dev_labels,
                            float** dev_predictions)
{
    // Flatten vector of vectors and push to arrays
    matrix2Cuda(features, dev_features);
    vector2Cuda(labels, dev_labels);

    // TODO
    // Check that preds is the correct shape and size
    float* preds = (float*)malloc(batch_size * sizeof(float));
    memset(preds, 0, batch_size * sizeof(float));
    CopyData(preds, batch_size, sizeof(float), dev_predictions);
}

template <class T>
void matrix2Cuda(std::vector<std::vector<T>>& m, T** dev_m)
{
	size_t m_rows = m.size();
	size_t m_cols = m[0].size();
	size_t m_total = m_rows * m_cols;
	T* arr = (T*)malloc(m_total * sizeof(T));
	for (size_t i = 0; i < m_rows; ++i)
        	for (size_t j = 0; j < m_cols; ++j)
            		arr[i*m_cols + j] = m[i][j];
    	assert(arr);
	CopyData(arr, m_total, sizeof(T), dev_m);
}

template <class T>
void vector2Cuda(std::vector<T>& v, T** dev_v)
{
        size_t v_total = v.size();
        T* arr = (T*)malloc(v_total * sizeof(T));
        for (size_t i = 0; i < v_total; ++i)
                        arr[i] = v[i];
        assert(arr);
        CopyData(arr, v_total, sizeof(T), dev_v);
}

template <class T>
void cuda2Matrix(T* dev_z, std::vector<std::vector<T>>& z)
{
    	size_t z_rows = z.size();
	size_t z_cols = z[0].size();
	size_t z_total = z_rows * z_cols;
	//creating an array to hold the data 
	T* arr = (T*)malloc(z_total * sizeof(T));

	//copying from GPU to CPU
    	checkCudaErrors(hipMemcpy(arr, dev_z, sizeof(T) * z_total, hipMemcpyDeviceToHost));
	
	//copying data into vector
	for (size_t i = 0; i < z_rows; ++i)
	{
                for (size_t j = 0; j < z_cols; ++j)
		{
                        z[i][j] = arr[i*z_cols + j];
			//std::cout << z[i][j] << " ";
		}
		//std::cout << std::endl;
	}
	//freeing memory
	free(arr);
}

template <class T>
void cuda2Vector(T** dev_z, std::vector<T>& z)
{
        size_t z_total = z.size();
        //creating an array to hold the data
        T* arr = (T*)malloc(z_total * sizeof(T));

        //copying from GPU to CPU
        checkCudaErrors(hipMemcpy(arr, dev_z, sizeof(T) * z_total, hipMemcpyDeviceToHost));

        //copying data into vector
        for (size_t i = 0; i < z_total; ++i)
	{
                        z[i] = arr[i];
			std::cout << arr[i];
	}
	std::cout << "\n";
        //freeing memory
        free(arr);
}

void cudaMatrixMultiply(std::vector<std::vector<float>> x,
                        std::vector<std::vector<float>> y,
                        std::vector<std::vector<float>>& z)
{
    float* dev_x;
    float* dev_y;
    float* dev_z;
    matrix2Cuda(x, &dev_x);
    matrix2Cuda(y, &dev_y);
    matrix2Cuda(z, &dev_z);
    //cuda_matrix_mult<<<x[0].size(), y.size()>>>(x[0].size(), y.size(), dev_x, dev_y, dev_z);
    size_t threads=32;
    
    //One block for each element in the result matrix
    dim3 dimGrid(x.size(), y[0].size(), 1);
    //Each block has 32 x 32 threads = 1024
    dim3 dimBlock(threads, 1, 1);
    //Using some shared memory, one float for each thread.
    unsigned int shared = threads * sizeof(float);
    cuda_matrix_mult<<<dimGrid, dimBlock, shared>>>(x[0].size(), y[0].size(), dev_x, dev_y, dev_z);
    //TODO
    cuda2Matrix(dev_z, z);
    //printf("We made it out\n");
}

void cudaMatrixMultiplyv2(std::vector<std::vector<float>> x,
                        std::vector<std::vector<float>> y,
                        std::vector<std::vector<float>>& z)
{
    float* dev_x;
    float* dev_y;
    float* dev_z;
    matrix2Cuda(x, &dev_x);
    matrix2Cuda(y, &dev_y);
    matrix2Cuda(z, &dev_z);
    //cuda_matrix_mult<<<x[0].size(), y.size()>>>(x[0].size(), y.size(), dev_x, dev_y, dev_z);
    size_t threads=y[0].size();
    //1024 is the max number of threads per block
    assert(threads <= 1024);

    //One block for each row in the resulting matrix
    dim3 dimGrid(x.size(), 1, 1);
    //Each block has as many threads as columns in the resulting matrix
    dim3 dimBlock(threads, 1, 1);
    //Using some shared memory, one float for each thread.
    unsigned int shared = threads * sizeof(float);
    cuda_matrix_mult_v2<<<dimGrid, dimBlock, shared>>>(x[0].size(), dev_x, dev_y, dev_z);
    //TODO
    cuda2Matrix(dev_z, z);
    //printf("We made it out\n");
}
